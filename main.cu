#include <iostream>
#include <vector>
#include <algorithm>
#include <chrono>
#include <iomanip>
#include <hip/hip_runtime.h>

using namespace std;

// Kernel CUDA para realizar o Counting Sort paralelamente
__global__ void countingSortKernel(int* arr, int* count, int* output, int n, int exp)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < n)
    {
        int digit = (arr[idx] / exp) % 10;
        atomicAdd(&count[digit], 1);
    }

    __syncthreads(); // Sincroniza os threads após o cálculo das contagens

    if (idx < 10)
    {
        for (int i = 1; i < 10; i++)
        {
            count[i] += count[i - 1];
        }
    }

    __syncthreads(); // Sincroniza antes de mover os elementos para a saída

    if (idx < n)
    {
        int digit = (arr[idx] / exp) % 10;
        int position = --count[digit];
        output[position] = arr[idx];
    }
}

// Função de Radix Sort com CUDA
void radixSortCuda(vector<int>& arr)
{
    int n = arr.size();
    int *d_arr, *d_count, *d_output;

    // Alocação de memória na GPU
    hipMalloc(&d_arr, n * sizeof(int));
    hipMalloc(&d_count, 10 * sizeof(int));
    hipMalloc(&d_output, n * sizeof(int));

    // Copiar dados para a GPU
    hipMemcpy(d_arr, arr.data(), n * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(d_count, 0, 10 * sizeof(int)); // Limpar o contador de dígitos

    int maxElem = *max_element(arr.begin(), arr.end());

    // Realiza Counting Sort para cada dígito
    for (int exp = 1; maxElem / exp > 0; exp *= 10)
    {
        // Lançamento de kernel para o Counting Sort
        int blockSize = 256;
        int numBlocks = (n + blockSize - 1) / blockSize;
        countingSortKernel<<<numBlocks, blockSize>>>(d_arr, d_count, d_output, n, exp);

        // Espera o kernel terminar
        hipDeviceSynchronize();

        // Copiar os resultados de volta para a memória do host
        hipMemcpy(arr.data(), d_output, n * sizeof(int), hipMemcpyDeviceToHost);

        // Limpar o contador de dígitos para o próximo dígito
        hipMemset(d_count, 0, 10 * sizeof(int));
    }

    // Liberação de memória na GPU
    hipFree(d_arr);
    hipFree(d_count);
    hipFree(d_output);
}

int main()
{
    vector<int> sizes = {100, 1000, 10000, 1000000, 10000000};

    // Itera sobre os diferentes tamanhos de entrada
    for (int n : sizes)
    {
        vector<int> arr(n);
        // Gera números aleatórios para o array
        for (int i = 0; i < n; i++)
        {
            arr[i] = rand() % 10000000;
        }

        // Inicia a medição de tempo
        auto start = chrono::high_resolution_clock::now();
        radixSortCuda(arr);
        auto end = chrono::high_resolution_clock::now();

        // Calcula a duração e imprime o tempo de execução
        chrono::duration<double> duration = end - start;
        cout << "Tempo de execucao Paralelo (CUDA) para " << n << " elementos: "
            << fixed << setprecision(6) << duration.count() << " segundos" << endl;
    }

    return 0;
}
